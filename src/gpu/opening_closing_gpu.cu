#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdio>
#include <numeric>

#include "detect_obj_gpu.hpp"
#include "helpers_gpu.hpp"

unsigned char *circular_kernel_gpu(int kernel_size)
{
    unsigned char *kernel = (unsigned char *)malloc(kernel_size * kernel_size
                                                    * sizeof(unsigned char));
    memset(kernel, 0, kernel_size * kernel_size);
    int radius = kernel_size / 2;
    for (int x = -radius; x < radius + 1; x++)
    {
        int y = (std::sqrt(radius * radius - (x * x)));
        for (int j = -y; j < y + 1; j++)
        {
            kernel[(j + radius) * kernel_size + (x + radius)] = 1;
            kernel[(-j + radius) * kernel_size + (x + radius)] = 1;
        }
    }
    size_t kernel_gpu_size = kernel_size * kernel_size * sizeof(unsigned char);
    unsigned char *kernel_gpu =
        cpyHostToDevice<unsigned char>(kernel, kernel_gpu_size);
    return kernel_gpu;
}

__global__ void perform_erosion_col_gpu(unsigned char *image, int rows,
                                        int cols, size_t kernel_size, int pitch)
{
    extern __shared__ int shared[];

    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    int start_k = kernel_size / 2;

    if (x >= cols || y >= rows)
        return;

    unsigned char sy = threadIdx.y;

    shared[sy + start_k] = image[y * pitch + x];

    // Add pading horizontal
    if (threadIdx.y == 0)
    {
        for (size_t i = 0; i < start_k; i++)
        {
            if ((y - start_k + i) >= 0 && (y - start_k + i) < rows)
                shared[i] = image[(y - start_k + i) * pitch + x];
            if ((y + i) >= 0 && (y + i) < rows)
                shared[blockDim.y - 1 + i] = image[(y + i) * pitch + x];
        }
    }

    __syncthreads();

    unsigned char res = 0;

    for (int i = -start_k; i < start_k + 1; i++)
    {
        int val = shared[sy + i];
        if (val != 0 && (res == 0 || val < res))
            res = val;
    }

    __syncthreads();
    image[y * pitch + x] = res;
}

__global__ void perform_erosion_line_gpu(unsigned char *image, int rows,
                                         int cols, size_t kernel_size,
                                         int pitch)
{
    extern __shared__ int shared[];

    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    int start_k = kernel_size / 2;

    if (x >= cols || y >= rows)
        return;

    unsigned char sx = threadIdx.x;

    shared[sx + start_k] = image[y * pitch + x];

    // Add pading horizontal
    if (threadIdx.x == 0)
    {
        for (size_t i = 0; i < start_k; i++)
        {
            if ((x - start_k + i) >= 0 && (x - start_k + i) < cols)
                shared[i] = image[y * pitch + x - start_k + i];
            if ((x + i) >= 0 && (x + i) < cols)
                shared[blockDim.x - 1 + i] = image[y * pitch + x + i];
        }
    }

    __syncthreads();

    unsigned char res = 0;

    for (int i = -start_k; i < start_k + 1; i++)
    {
        int val = shared[sx + i];
        if (val != 0 && (res == 0 || val < res))
            res = val;
    }

    __syncthreads();
    image[y * pitch + x] = res;
}

__global__ void perform_dilation_line_gpu(unsigned char *image, int rows,
                                          int cols, size_t kernel_size,
                                          int pitch)
{
    extern __shared__ int shared[];

    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    int start_k = kernel_size / 2;

    if (x >= cols || y >= rows)
        return;

    unsigned char sx = threadIdx.x;

    shared[sx + start_k] = image[y * pitch + x];

    // Add pading horizontal
    if (threadIdx.x == 0)
    {
        for (size_t i = 0; i < start_k; i++)
        {
            if ((x - start_k + i) >= 0 && (x - start_k + i) < cols)
                shared[i] = image[y * pitch + x - start_k + i];
            if ((x + i) >= 0 && (x + i) < cols)
                shared[blockDim.x - 1 + i] = image[y * pitch + x + i];
        }
    }

    __syncthreads();

    unsigned char res = 0;

    for (int i = -start_k; i < start_k + 1; i++)
    {
        int val = shared[sx + i];
        if (val > res)
            res = val;
    }

    __syncthreads();
    image[y * pitch + x] = res;
}

__global__ void perform_dilation_col_gpu(unsigned char *image, int rows,
                                         int cols, size_t kernel_size,
                                         int pitch)
{
    extern __shared__ int shared[];

    int pos = blockDim.x * blockIdx.x + threadIdx.x;

    if (pos >= cols * rows)
        return;
    int start_k = kernel_size / 2;

    int y = pos / cols;
    int x = pos % cols;

    unsigned char sx = threadIdx.x;

    shared[sx + start_k] = image[y * pitch + x];

    // Add pading horizontal
    if (sx == 0)
    {
        for (size_t i = 0; i < start_k; i++)
        {
            if ((x - start_k + i) >= 0 && (x - start_k + i) < rows)
                shared[i] = image[(y - start_k + i) * pitch + x];
            if ((y + i) >= 0 && (y + i) < rows)
                shared[blockDim.y - 1 + i] = image[(y + i) * pitch + x];
        }
    }

    __syncthreads();

    unsigned char res = 0;

    for (int i = -start_k; i < start_k + 1; i++)
    {
        int val = shared[sy + i];
        if (val > res)
            res = val;
    }

    __syncthreads();
    image[y * pitch + x] = res;
}

void erosion_gpu(unsigned char *obj, size_t rows, size_t cols, size_t k_size,
                 unsigned char *kernel, size_t pitch, int thx, int thy)
{
    const int threads = 1024;
    const int size_shared = threads + k_size;
    const int blocks = std::ceil(float(cols * rows) / float(threads.x));

    perform_erosion_gpu<<<blocks, threads, size_shared>>>(obj, rows, cols,
                                                          k_size, pitch);
    cudaCheckError();
    hipDeviceSynchronize();
}

void dilation_gpu(unsigned char *obj, size_t rows, size_t cols, size_t k_size,
                  unsigned char *kernel, size_t pitch, int thx, int thy)
{
    const int threads = 1024;
    const int size_shared = threads + k_size;
    const int blocks = std::ceil(float(cols * rows) / float(threads.x));

    perform_dilation_gpu<<<blocks, threads, size_shared>>>(obj, rows, cols,
                                                           k_size, pitch);
    cudaCheckError();
    hipDeviceSynchronize();
}

/*
unsigned char *circular_kernel_gpu(int kernel_size)
{
    unsigned char *kernel = (unsigned char *)malloc(kernel_size * kernel_size
                                                    * sizeof(unsigned char));
    memset(kernel, 0, kernel_size * kernel_size);
    int radius = kernel_size / 2;
    for (int x = -radius; x < radius + 1; x++)
    {
        int y = (std::sqrt(radius * radius - (x * x)));
        for (int j = -y; j < y + 1; j++)
        {
            kernel[(j + radius) * kernel_size + (x + radius)] = 1;
            kernel[(-j + radius) * kernel_size + (x + radius)] = 1;
        }
    }
    size_t kernel_gpu_size = kernel_size * kernel_size * sizeof(unsigned char);
    unsigned char *kernel_gpu =
        cpyHostToDevice<unsigned char>(kernel, kernel_gpu_size);
    return kernel_gpu;
}

__global__ void perform_dilation_gpu(unsigned char *image, int rows, int cols,
                                     size_t kernel_size, unsigned char *kernel,
                                     int pitch)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= cols || y >= rows)
        return;

    int start_k = kernel_size / 2;

    unsigned char res = 0;

    for (int i = -start_k; i < start_k + 1; i++)
    {
        for (int j = -start_k; j < start_k + 1; j++)
        {
            if ((y + j) >= 0 && (y + j) < rows && (x + i) >= 0
                && (x + i) < cols)
            {
                int mult = image[(y + j) * pitch + (x + i)]
                    * kernel[(j + start_k) * kernel_size + (i + start_k)];
                if (mult > res)
                    res = mult;
            }
        }
    }

    __syncthreads();

    image[y * pitch + x] = res;
}

__global__ void perform_erosion_gpu(unsigned char *image, int rows, int cols,
                                    size_t kernel_size, unsigned char *kernel,
                                    int pitch)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= cols || y >= rows)
        return;

    int start_k = kernel_size / 2;

    unsigned char res = 0;

    for (int i = -start_k; i < start_k + 1; i++)
    {
        for (int j = -start_k; j < start_k + 1; j++)
        {
            if ((y + j) >= 0 && (y + j) < rows && (x + i) >= 0
                && (x + i) < cols)
            {
                int mult = image[(y + j) * pitch + (x + i)]
                    * kernel[(j + start_k) * kernel_size + (i + start_k)];
                if (mult != 0 && (res == 0 || mult < res))
                    res = mult;
            }
        }
    }

    __syncthreads();

    image[y * pitch + x] = res;
}


void erosion_gpu(unsigned char *obj, size_t rows, size_t cols, size_t k_size,
                 unsigned char *kernel, size_t pitch, int thx, int thy) {
    const dim3 threads(thx, thy);
    const dim3 blocks(std::ceil(float(cols) / float(threads.x)),
std::ceil(float(rows) / float(threads.y)));

    perform_erosion_gpu<<<blocks, threads>>>(obj, rows, cols, k_size, kernel,
pitch); cudaCheckError(); hipDeviceSynchronize();
}

void dilation_gpu(unsigned char *obj, size_t rows, size_t cols, size_t k_size,
                 unsigned char *kernel, size_t pitch, int thx, int thy) {
    const dim3 threads(thx, thy);
    const dim3 blocks(std::ceil(float(cols) / float(threads.x)),
std::ceil(float(rows) / float(threads.y)));

    perform_dilation_gpu<<<blocks, threads>>>(obj, rows, cols, k_size, kernel,
pitch); cudaCheckError(); hipDeviceSynchronize();
}
*/
/*

__global__ void perform_dilation_gpu(unsigned char *image, int rows, int cols,
                                     size_t kernel_size, int pitch)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= cols || y >= rows)
        return;

    int radius = kernel_size / 2;
    int res = 0;

    for (int i = -radius; i < radius + 1; i++)
    {
        int k_y = sqrt(float(radius * radius - (i * i)));
        for (int j = -k_y; j < k_y + 1; j++)
        {
            if ((y + j) >= 0 && (y + j) < rows && (x + i) >= 0
                && (x + i) < cols)
            {
                int mult = image[(y + j) * pitch + (x + i)];
                if (mult > res)
                    res = mult;

                mult = image[(y - j) * pitch + (x + i)];
                if (mult > res)
                    res = mult;
            }
        }
    }

    __syncthreads();

    image[y * pitch + x] = res;
}

__global__ void perform_erosion_gpu(unsigned char *image, int rows, int cols,
                                    size_t kernel_size, int pitch)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= cols || y >= rows)
        return;

    int radius = kernel_size / 2;
    int res = 0;

    for (int i = -radius; i < radius + 1; i++)
    {
        int k_y = sqrt(float(radius * radius - (i * i)));
        for (int j = -k_y; j < k_y + 1; j++)
        {
            if ((y + j) >= 0 && (y + j) < rows && (x + i) >= 0
                && (x + i) < cols)
            {
                int mult = image[(y + j) * pitch + (x + i)];
                if (res == 0 || mult < res)
                    res = mult;

                mult = image[(y - j) * pitch + (x + i)];
                if (res == 0 || mult < res)
                    res = mult;
            }
        }
    }

    __syncthreads();

    image[y * pitch + x] = res;
}
*/
